#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <assert.h>

//Include external files
#include "../headers/errormessage.h"

//forward declaration
__global__ void vecScale(float *array, float scale, int n);

extern "C" void __device__fft_gather(int fft_type, float scale, float *data, hipfftComplex *dataw, int ntw, int nttr, int nt) {
	int rank = 1, inembed = { 0 }, onembed = { 0 }, istride = 1, idist = nt, ostride = 1, odist = ntw;
	int iDev = 0;  //only one device at the moment...
	int blockSize;   // The launch configurator returned block size
	int minGridSize; // The minimum grid size needed to achieve the
	                 // maximum occupancy for a full device launch
	int gridSize;    // The actual grid size needed, based on input size
	int arrayCount = nt*nttr;

	hipfftHandle planfwcu = { 0 }, planbwcu = { 0 };
	hipfftReal *hostInputData, *deviceInputData,*deviceOutputData;
	hipfftComplex *hostInputDataw, *deviceInputDataw, *deviceOutputDataw;
	hipError_t error_id;

	// For 1D transform:   input[ b * idist + x * istride]
	//                    output[ b * odist + x * ostride]

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, iDev);

	if (fft_type == HIPFFT_FORWARD) {
		//****************Host methods***********
		//Host side Input data allocation
		hostInputData = (hipfftReal*) data; //pointer to the original loaded data

		//Host side output data allocation
		//hipfftComplex *hostOutputData = (hipfftComplex*)malloc(ntw * nttr * sizeof(hipfftComplex));

		//****************Device methods***********
		//Device side input data allocation and initialization
		CHECK_CUDA_ERRORS(hipMalloc((void** )&deviceInputData,nt * nttr * sizeof(hipfftReal)));
		//Some later adaptation here to make the memory copy from the Host to Device to a chunk size
		CHECK_CUDA_ERRORS(hipMemcpy(deviceInputData, hostInputData,nt * nttr * sizeof(hipfftReal), hipMemcpyHostToDevice));
		//Device side output data allocation
		CHECK_CUDA_ERRORS(hipMalloc((void** )&deviceOutputDataw,ntw * nttr * sizeof(hipfftComplex)));

		//Make a plan
		CHECK_CUFFT_ERRORS(hipfftPlanMany(&planfwcu, rank, &nt, &inembed, istride, idist,&onembed, ostride, odist, HIPFFT_R2C, nttr));

		fprintf(stderr, "CUFFT FORWARD\n");
		CHECK_CUFFT_ERRORS(hipfftExecR2C(planfwcu, deviceInputData, deviceOutputDataw));

		hipDeviceSynchronize(); //kernel is guaranteed to finish
		//Can do printing here...

		//Device->Host copy of the results
		CHECK_CUDA_ERRORS(hipMemcpy(dataw, deviceOutputDataw,ntw * nttr * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

		//Destroy the plan
		CHECK_CUFFT_ERRORS(hipfftDestroy(planfwcu));

		//Release device memory
		CHECK_CUDA_ERRORS(hipFree(deviceInputData));
		CHECK_CUDA_ERRORS(hipFree(deviceOutputDataw));

	} else if (fft_type == HIPFFT_BACKWARD) {
		//Device properties:
		//fprintf(stderr,"  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
	    //fprintf(stderr,"Warp size:                                       %d\n",  deviceProp.warpSize);
	    //fprintf(stderr,"Maximum # Threads per MultiProcessor:            %d\n",deviceProp.maxThreadsPerMultiProcessor);

		//redefinition of stride I/O for inverse FFT plan
		idist = ntw, odist = nt;

		//****************Host methods***********
		//Host side Input data allocation
		hostInputDataw = dataw;    //pointer to the original loaded data

		//Host side output data allocation
		hipfftReal *hostOutputData = (hipfftReal*)data;
		//hostOutputData = (hipfftReal*) malloc(nt * nttr * sizeof(hipfftReal));

		//****************Device methods***********
		//Device side input data allocation and initialization
		CHECK_CUDA_ERRORS(hipMalloc((void** )&deviceInputDataw,ntw * nttr * sizeof(hipfftComplex)));

		//Some later adaptation here to make the memory copy from the Host to Device to a chunk size
		// check for error
		CHECK_CUDA_ERRORS(hipMemcpy(deviceInputDataw, hostInputDataw,ntw * nttr * sizeof(hipfftComplex),hipMemcpyHostToDevice));

		//Device side output data allocation
		CHECK_CUDA_ERRORS(hipMalloc((void** )&deviceOutputData,nt * nttr * sizeof(hipfftReal)));

		//make a plan using CUFFT
		//CHECK_CUFFT_ERRORS(hipfftPlanMany(&planbwcu, rank, &nt, &inembed, istride, idist, &onembed, ostride, odist, HIPFFT_C2R, nttr)) This is not working
		CHECK_CUFFT_ERRORS(hipfftPlanMany(&planbwcu, rank, &nt, &onembed, istride, idist,&onembed, ostride, odist, HIPFFT_C2R, nttr));

		//fprintf(stderr, "CUFFT INVERSE\n");

		CHECK_CUFFT_ERRORS(hipfftExecC2R(planbwcu, deviceInputDataw, deviceOutputData));

		//SCALE THE OUTPUT DATA BY 1/NT
		//Find out the maximum occupancy
		hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,vecScale, 0, 0);

		// Round up according to array size
		gridSize = (arrayCount + blockSize - 1) / blockSize;

		//fprintf(stderr,"gridSize=%d blockSize=%d \n",gridSize,blockSize);

		// calculate theoretical occupancy
		int maxActiveBlocks;
		hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks,vecScale, blockSize,0);
		/*float occupancy = (maxActiveBlocks * blockSize / deviceProp.warpSize) /
						(float)(deviceProp.maxThreadsPerMultiProcessor /
								deviceProp.warpSize); */

		//fprintf(stderr,"maxActiveBlocks=%d Launched blocks of size %d. Theoretical occupancy: %f\n", maxActiveBlocks, blockSize, occupancy);

		//Launch the scaling Kernel
		vecScale<<<gridSize, blockSize>>>(deviceOutputData, scale, arrayCount);
		error_id=hipGetLastError();
		if (error_id != hipSuccess) {
			fprintf(stderr,"GPU kernel assert: %s %s %d\n", hipGetErrorString(error_id), __FILE__, __LINE__);
			assert(0);
		}

		CHECK_CUDA_ERRORS(hipDeviceSynchronize());  //kernel is guaranteed to finish
		//Can do printing here...

		//Device->Host copy of the results
		CHECK_CUDA_ERRORS(hipMemcpy(hostOutputData, deviceOutputData,nt * nttr * sizeof(hipfftReal), hipMemcpyDeviceToHost));

		data = (float*)hostOutputData;

		//Destroy the plan
		hipfftDestroy(planbwcu);

		//Release device memory
		CHECK_CUDA_ERRORS(hipFree(deviceInputDataw));
		CHECK_CUDA_ERRORS(hipFree(deviceOutputData));
	}

	//Release Host memory
	//hipFree(hostOutputData);

}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecScale(float *array, float scale, int n) {
	// Get our global thread ID
	int idx = blockIdx.x*blockDim.x + threadIdx.x; //blockId.x: which block is it?
												   //blockDim.x: How many threads there are in a block?
	 	 	 	 	 	 	 	 	 	 	 	   //threadId.x: Which thread it is inside that block?

	 // Make sure we do not go out of bounds
	 if (idx < n)
		 array[idx] *= scale;
}

static const char *_cudaGetErrorEnum(hipfftResult error) {
	switch (error) {
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";

	case HIPFFT_INCOMPLETE_PARAMETER_LIST:
		return "HIPFFT_INCOMPLETE_PARAMETER_LIST";

	case HIPFFT_INVALID_DEVICE:
		return "HIPFFT_INVALID_DEVICE";

	case HIPFFT_PARSE_ERROR:
		return "HIPFFT_PARSE_ERROR";

	case HIPFFT_NO_WORKSPACE:
		return "HIPFFT_NO_WORKSPACE";

	case HIPFFT_NOT_IMPLEMENTED:
		return "HIPFFT_NOT_IMPLEMENTED";

	case CUFFT_LICENSE_ERROR:
		return "CUFFT_LICENSE_ERROR";

	case HIPFFT_NOT_SUPPORTED:
		return "HIPFFT_NOT_SUPPORTED";

	}

	return "<unknown>";
}
